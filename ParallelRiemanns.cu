#include "hip/hip_runtime.h"
//Christian Douglas
//Joseph Williamson
//Alex Trampert
//Riemann Summ Group Project
//Parallel code

#include <stdio.h>

#define NUMBER_OF_THREADS 1024

// CUDA kernel
__global__ void findArea(float* deltaX)
{   __shared__float sum = 0;

    // Get our global thread ID
    float id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id == 0 || id == NUMBER_OF_THREADS) {
        sum += (id * id) + (id * 3) + 3;
    }  else {
        float x = (id) * deltaX;
        sum += 2 * ((x * x) + (x * 3) + 3);
    }
}

int main()
{
    //host variables
    float a = 0, b = 100,
     deltaX = (b-a)/NUMBER_OF_THREADS;
    //device variables
    float* deltaX_dev;

    //allocate on gpu 
    hipMalloc((void**)a_dev, deltaX);
    hipMemcpy(deltaX_dev, deltaX, deltaX, hipMemcpyHostToDevice);

    dim3 GridDim(1,1);
    dim3 BlockDim(NUMBER_OF_THREADS,1);

    //call kernel 
    findArea<<<GridDim,BlockDim>>>(deltaX_dev);
    
    hipFree(deltaX_dev);
    
    printf("Riemann's Sum: %f\n", sum);

    return 0;
}